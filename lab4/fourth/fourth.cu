#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>


#define CHECK(call) { \
    const hipError_t error = call; \
    if (error != hipSuccess) { \
        fprintf(stderr, "CUDA Error: %s:%d, %s\n", __FILE__, __LINE__, hipGetErrorString(error)); \
        exit(1); \
    } \
}

__global__ void addKernel(float *a, float *b, float *result, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) result[idx] = a[idx] + b[idx];
}

__global__ void subtractKernel(float *a, float *b, float *result, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) result[idx] = a[idx] - b[idx];
}

__global__ void multiplyKernel(float *a, float *b, float *result, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) result[idx] = a[idx] * b[idx];
}

__global__ void divideKernel(float *a, float *b, float *result, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) result[idx] = a[idx] / b[idx];
}

void sequentialOperations(float *a, float *b, float *result, int size, char op) {
    for (int i = 0; i < size; i++) {
        switch (op) {
            case '+': result[i] = a[i] + b[i]; break;
            case '-': result[i] = a[i] - b[i]; break;
            case '*': result[i] = a[i] * b[i]; break;
            case '/': result[i] = a[i] / b[i]; break;
        }
    }
}

int main(int argc, char **argv) {
    const int rows = 316;
    const int cols = 316;
    const int size = rows * cols;
    const int bytes = size * sizeof(float);

    float *a = (float *)malloc(bytes);
    float *b = (float *)malloc(bytes);
    float *result_seq = (float *)malloc(bytes);
    float *result_par = (float *)malloc(bytes);

    srand(time(NULL));
    for (int i = 0; i < size; i++) {
        a[i] = (float)(rand() % 100 + 1);
        b[i] = (float)(rand() % 100 + 1);
    }

    float *d_a, *d_b, *d_result;
    CHECK(hipMalloc(&d_a, bytes));
    CHECK(hipMalloc(&d_b, bytes));
    CHECK(hipMalloc(&d_result, bytes));
    CHECK(hipMemcpy(d_a, a, bytes, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_b, b, bytes, hipMemcpyHostToDevice));

    dim3 block(256);
    dim3 grid((size + block.x - 1) / block.x);

    // Parallel
    hipEvent_t start, stop;
    float elapsed;
    CHECK(hipEventCreate(&start));
    CHECK(hipEventCreate(&stop));

    CHECK(hipEventRecord(start));
    addKernel<<<grid, block>>>(d_a, d_b, d_result, size);
    subtractKernel<<<grid, block>>>(d_a, d_b, d_result, size);
    multiplyKernel<<<grid, block>>>(d_a, d_b, d_result, size);
    divideKernel<<<grid, block>>>(d_a, d_b, d_result, size);
    CHECK(hipEventRecord(stop));
    CHECK(hipEventSynchronize(stop));
    CHECK(hipEventElapsedTime(&elapsed, start, stop));
    printf("Parallel time: %.5f seconds\n", elapsed / 1000.0f);

    CHECK(hipMemcpy(result_par, d_result, bytes, hipMemcpyDeviceToHost));

    // Sequential
    clock_t seq_start = clock();
    sequentialOperations(a, b, result_seq, size, '+');
    sequentialOperations(a, b, result_seq, size, '-');
    sequentialOperations(a, b, result_seq, size, '*');
    sequentialOperations(a, b, result_seq, size, '/');
    clock_t seq_end = clock();
    float seq_time = (float)(seq_end - seq_start) / CLOCKS_PER_SEC;
    printf("Sequential time: %.5f seconds\n", seq_time);

    free(a); free(b); free(result_seq); free(result_par);
    CHECK(hipFree(d_a)); CHECK(hipFree(d_b)); CHECK(hipFree(d_result));
    CHECK(hipEventDestroy(start)); CHECK(hipEventDestroy(stop));

    return 0;
}
